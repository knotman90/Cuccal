#include "hip/hip_runtime.h"
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
//#include "CA_GPU.cuh"

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#include "CA_GPU.cuh"




/*STATIC AND UTILITY CA FUNCTIONS*/
__device__  inline unsigned int  CA_GPU::d_mod (int m, int n){
	return m >= 0 ? m % n : ( n - abs( m%n ) ) % n;
}


__device__  inline   unsigned  int CA_GPU::d_getLinearIndexToroidal(unsigned int i, unsigned int j,unsigned int rows, unsigned int cols){
	return (this->d_mod(i,rows)*cols+this->d_mod(j,cols));
}

__device__ inline   unsigned  int CA_GPU::d_getLinearIndexNormal(unsigned int i, unsigned int j, unsigned int rows,unsigned int cols){
	return (i*cols+j);
}

__device__  inline unsigned   int CA_GPU::d_getLinearIndexToroidalLinear(unsigned int index,unsigned int rows, unsigned int cols){
	return d_mod(index,rows*cols);
}


__device__ unsigned int CA_GPU::getNeighborIndex_MOORE_Toroidal(unsigned int i, unsigned int j,unsigned int neighbor,unsigned int rows, unsigned int cols){
	switch(neighbor){
	case 0:
		return d_getLinearIndexToroidal(i,j,rows,cols);
	case 1:
		return d_getLinearIndexToroidal(i-1,j,rows,cols);//one row up
	case 2:
		return d_getLinearIndexToroidal(i,j-1,rows,cols);//same row one coloumn left
	case 3:
		return d_getLinearIndexToroidal(i,j+1,rows,cols);//same row one coloumn right
	case 4:
		return d_getLinearIndexToroidal(i+1,j,rows,cols);//same column one row down
	case 5:
		return d_getLinearIndexToroidal(i-1,j-1,rows,cols);//one row up one col left
	case 6:
		return d_getLinearIndexToroidal(i+1,j-1,rows,cols);//one row down one col left
	case 7:
		return d_getLinearIndexToroidal(i+1,j+1,rows,cols);//row down col right
	case 8:
		return d_getLinearIndexToroidal(i-1,j+1,rows,cols);//row up col right
	}

	return NULL;//it should never be executed
}








/* ------------------START GET SUBSTATE FAMILY FUNCTION------------------*/
__device__  bool CA_GPU::getSubstateValue_BOOL(unsigned int substateLabel,unsigned int i, unsigned int j) const{
	return ((bool*)d_substates[substateLabel])[getLinearIndex(i,j,scalars->rows,scalars->cols)];
}

__device__ double CA_GPU::getSubstateValue_DOUBLE(unsigned int substateLabel,unsigned int i, unsigned int j)const{

	return ((double*) d_substates[substateLabel])[getLinearIndex(i,j, scalars->rows, scalars->cols)];
}

__device__ float CA_GPU::getSubstateValue_FLOAT(unsigned int substateLabel,unsigned int i, unsigned int j)const{
	return ((float*) d_substates[substateLabel])[getLinearIndex(i,j, scalars->rows, scalars->cols)];
}

__device__ int CA_GPU::getSubstateValue_INT(unsigned int substateLabel,unsigned int i, unsigned int j)const{
	return ((int*) d_substates[substateLabel])[this->getLinearIndex(i,j, scalars->rows, scalars->cols)];
}

__device__ char CA_GPU::getSubstateValue_CHAR(unsigned int substateLabel,unsigned int i, unsigned int j)const{
	return ((char*) d_substates[substateLabel])[getLinearIndex(i,j, scalars->rows, scalars->cols)];
}

//mono index cell representation
__device__ bool CA_GPU::getSubstateValue_BOOL(unsigned int substateLabel,unsigned int index) const{
	return ((bool*) d_substates[substateLabel])[index];
}

__device__ double CA_GPU::getSubstateValue_DOUBLE(unsigned int substateLabel,unsigned int index)const{
	return ((double*) d_substates[substateLabel])[index];
}

__device__ float CA_GPU::getSubstateValue_FLOAT(unsigned int substateLabel,unsigned int index)const{
	return ((float*) d_substates[substateLabel])[index];
}

__device__ int CA_GPU::getSubstateValue_INT(unsigned int substateLabel,unsigned int index)const{
	return ((int*) d_substates[substateLabel])[index];
}

__device__ char CA_GPU::getSubstateValue_CHAR(unsigned int substateLabel,unsigned int index)const{
	return ((char*) d_substates[substateLabel])[index];
}




/* ------------------END GET SUBSTATE VALUE FAMILY------------------*/


/* ----------------START SET SUBSTATE FAMILY FUNCTION ------------------*/
__device__ void CA_GPU::setSubstateValue_BOOL(unsigned int substateLabel,unsigned int i, unsigned int j,bool const value) {
	((bool*) d_substates[substateLabel])[getLinearIndex(i,j, scalars->rows, scalars->cols)]=value;
}

__device__ void CA_GPU::setSubstateValue_DOUBLE(unsigned int substateLabel,unsigned int i, unsigned int j, double const value){
	((double*) d_substates[substateLabel])[getLinearIndex(i,j, scalars->rows, scalars->cols)]=value;
}

__device__ void CA_GPU::setSubstateValue_FLOAT(unsigned int substateLabel,unsigned int i, unsigned int j,float const value){
	((float*) d_substates[substateLabel])[getLinearIndex(i,j, scalars->rows, scalars->cols)]=value;
}

__device__ void CA_GPU::setSubstateValue_INT(unsigned int substateLabel,unsigned int i, unsigned int j,int const value){
	((int*) d_substates[substateLabel])[getLinearIndex(i,j, scalars->rows, scalars->cols)]=value;
}

__device__ void CA_GPU::setSubstateValue_CHAR(unsigned int substateLabel,unsigned int i, unsigned int j,char const value){
	((char*) d_substates[substateLabel])[getLinearIndex(i,j, scalars->rows, scalars->cols)]=value;
}


__device__ void CA_GPU::setSubstateValue_BOOL(unsigned int substateLabel,unsigned int index,bool const value) {
	((bool*) d_substates[substateLabel])[index]=value;
}

__device__ void CA_GPU::setSubstateValue_DOUBLE(unsigned int substateLabel,unsigned int index, double const value){
	((double*) d_substates[substateLabel])[index]=value;
}

__device__ void CA_GPU::setSubstateValue_FLOAT(unsigned int substateLabel,unsigned int index,float const value){
	((float*) d_substates[substateLabel])[index]=value;
}

__device__ void CA_GPU::setSubstateValue_INT(unsigned int substateLabel,unsigned int index,int const value){
	((int*) d_substates[substateLabel])[index]=value;
}

__device__ void CA_GPU::setSubstateValue_CHAR(unsigned int substateLabel,unsigned int index,char const value){
	((char*) d_substates[substateLabel])[index]=value;
}


/* ------------------END SET SUBSTATE VALUE FAMILY------------------*/



__device__ unsigned int CA_GPU::getLinearIndex(unsigned int i, unsigned int j,unsigned int rows, unsigned int cols)const{
	return (i*cols+j);
}



