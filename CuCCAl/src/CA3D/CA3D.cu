#include "hip/hip_runtime.h"
/*
 * CA.cpp
 *
 *  Created on: 21/mar/2014
 *      Author: davide
 */

#include "CA3D.cuh"

#define CUDA_CHECK_RETURN(value) {											\
		hipError_t _m_cudaStat = value;										\
		if (_m_cudaStat != hipSuccess) {										\
			fprintf(stderr, "Error %s at line %d in file %s\n",					\
					hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
					exit(1);															\
		} }

//numcells=yDim*xDim in constructor CA
void* CA3D::allocateGPUBuffer(void * d_buffer,TYPE type){
	switch(type){

	case FLOAT:
		CUDA_CHECK_RETURN(hipMalloc((void**)&d_buffer,sizeof(float)*numCells));
		break;
	case DOUBLE:
		CUDA_CHECK_RETURN(hipMalloc((void**)&d_buffer,sizeof(double)*numCells));
		break;
	case CHAR:
		CUDA_CHECK_RETURN(hipMalloc((void**)&d_buffer,sizeof(char)*numCells));
		break;
	case INT:
		CUDA_CHECK_RETURN(hipMalloc((void**)&d_buffer,sizeof(int)*numCells));
		break;
	case BOOL:
		CUDA_CHECK_RETURN(hipMalloc((void**)&d_buffer,sizeof(bool)*numCells));
		break;

	}

	return d_buffer;
}


void CA3D::copyBufferFromGPU(void* h_to, void* d_from, TYPE type){

	switch(type){

	case FLOAT:
		CUDA_CHECK_RETURN(hipMemcpy(h_to,d_from,sizeof(float)*numCells,hipMemcpyDeviceToHost));
		break;
	case DOUBLE:
		CUDA_CHECK_RETURN(hipMemcpy(h_to,d_from,sizeof(double)*numCells,hipMemcpyDeviceToHost));
		break;
	case CHAR:
		CUDA_CHECK_RETURN(hipMemcpy(h_to,d_from,sizeof(char)*numCells,hipMemcpyDeviceToHost));
		break;
	case INT:
		CUDA_CHECK_RETURN(hipMemcpy(h_to,d_from,sizeof(int)*numCells,hipMemcpyDeviceToHost));
		break;
	case BOOL:
		CUDA_CHECK_RETURN(hipMemcpy(h_to,d_from,sizeof(bool)*numCells,hipMemcpyDeviceToHost));
		break;

	}
}

void CA3D::copyBufferToGPU(void* d_to, void* h_from, TYPE type){

	switch(type){

	case FLOAT:
		CUDA_CHECK_RETURN(hipMemcpy(d_to,h_from,sizeof(float)*numCells,hipMemcpyHostToDevice));
		break;
	case DOUBLE:
		CUDA_CHECK_RETURN(hipMemcpy(d_to,h_from,sizeof(double)*numCells,hipMemcpyHostToDevice));
		break;
	case CHAR:
		CUDA_CHECK_RETURN(hipMemcpy(d_to,h_from,sizeof(char)*numCells,hipMemcpyHostToDevice));
		break;
	case INT:
		CUDA_CHECK_RETURN(hipMemcpy(d_to,h_from,sizeof(int)*numCells,hipMemcpyHostToDevice));
		break;
	case BOOL:
		CUDA_CHECK_RETURN(hipMemcpy(d_to,h_from,sizeof(bool)*numCells,hipMemcpyHostToDevice));
		break;

	}
}

__global__ void initializeDCA(void** d_AllocatedpointerSubstates,TYPE* d_substateTypes,SCALARS_CA_GPU3D* scalarsTOCPY,CA_GPU3D* d_CA){

	d_CA->d_substates=d_AllocatedpointerSubstates;
	d_CA->d_substateTypes=d_substateTypes;
	d_CA->scalars=scalarsTOCPY;


}

__global__ void printValues(CA_GPU3D* d_CA){

	printf("SCALARS\n yDim=%i, xDim=%i , zDim=%i\n ",d_CA->scalars->yDim,d_CA->scalars->xDim,d_CA->scalars->zDim);


}

void CA3D::initializeGPUAutomata(){
	//allocate GPU_CA on GPU
	CUDA_CHECK_RETURN(hipMalloc(&d_CA,sizeof(CA_GPU3D)));
	d_CA_TOCOPY= new CA_GPU3D();


	//allocate memory ON GPU

	/*allocate all the substates ON GPU
	substate_size=substate_count=real number of registered buffers -> coherent state of the automata
	checked befor of GPU initialization
	conversion between unsigned int(substateTypes) and TYPE is legal*/
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_CA_TOCOPY->d_substates,sizeof(void*)*substates_size));

	d_subPointer = (void**)malloc(sizeof(void*)*substates_size);
	for(int i=0;i<substates_size;i++){
		d_subPointer[i]=allocateGPUBuffer(d_subPointer[i],(TYPE)substateTypes[i]);
		copyBufferToGPU(d_subPointer[i],substates[i],(TYPE)substateTypes[i]);

	}
	CUDA_CHECK_RETURN(hipMemcpy(d_CA_TOCOPY->d_substates,d_subPointer,sizeof(void*)*substates_size,hipMemcpyHostToDevice));
	//CUDA_CHECK_RETURN(hipFree((void*)(&d_CA_TOCOPY->d_substates[1])));

	//substates type array (allocation and copy, that's a constant array usually)
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_CA_TOCOPY->d_substateTypes,sizeof(TYPE)*substates_size));
	CUDA_CHECK_RETURN(hipMemcpy(d_CA_TOCOPY->d_substateTypes,substateTypes,sizeof(TYPE)*substates_size,hipMemcpyHostToDevice));

	//copyScalars. First create structure to be copied, then allocate memory on GPU->copy structure on GPU->
	//->then link d_CA_TOCOPY->scalars to d_CA->scalars whithin a kernel
	SCALARS_CA_GPU3D* scalars_TOPCOPY = new SCALARS_CA_GPU3D();
	scalars_TOPCOPY->xDim=xDim;
	scalars_TOPCOPY->yDim=yDim;
	scalars_TOPCOPY->zDim=zDim;
	scalars_TOPCOPY->stop=stop;
	scalars_TOPCOPY->steps=steps;
	scalars_TOPCOPY->isToroidal=isToroidal;
	scalars_TOPCOPY->substates_size=substates_size;
	scalars_TOPCOPY->numCells=numCells;

	CUDA_CHECK_RETURN(hipMalloc((void**)&d_CA_TOCOPY->scalars,sizeof(SCALARS_CA_GPU3D)));
	CUDA_CHECK_RETURN(hipMemcpy(d_CA_TOCOPY->scalars,scalars_TOPCOPY,sizeof(SCALARS_CA_GPU3D),hipMemcpyHostToDevice));

	free(scalars_TOPCOPY);//not needed anymore

	hipDeviceSynchronize();
	initializeDCA<<<1,1>>>(d_CA_TOCOPY->d_substates,d_CA_TOCOPY->d_substateTypes,d_CA_TOCOPY->scalars,d_CA);


	hipDeviceSynchronize();
	printValues<<<1,10>>>(d_CA);
	hipDeviceSynchronize();



	printDebug("GPU memory allocated");
}

void CA3D::cleanUpGPUAutomata(){
	//FREE MEMORY ON GPU-> remember to free first all the buffers INSIDE the struct
	printDebug("inizio cleanUP GPU");
	for(int i=0;i<substates_size;i++){
		CUDA_CHECK_RETURN(hipFree((void*)((d_subPointer[i]))));

	}

	//free scalars GPU
	CUDA_CHECK_RETURN(hipFree((void*)d_CA_TOCOPY->scalars));
	//CUDA_CHECK_RETURN(hipFree(d_CA));
	free(d_subPointer);
	printDebug("GPU memory freeed");
}

unsigned long long int CA3D::getSteps() const{
	return steps;
}

unsigned int CA3D::getToroidalLinearIndex(unsigned int linearIndex){
	return hd_mod(linearIndex,yDim*xDim);
}

int CA3D::loadSubstate(SUBSTATE_LABEL substateLabel, const char* const pathToFile){
	short int status =SUCCESS_OPENING_FILE;
	unsigned int type= substateTypes[substateLabel];
	switch(type){
	case FLOAT:
		status=CA_load_substate_FILE3D(pathToFile,(float*)(substates[substateLabel]),yDim,xDim,zDim);
		break;
	case DOUBLE:
		status=CA_load_substate_FILE3D(pathToFile,(double*)(substates[substateLabel]),yDim,xDim,zDim);
		break;
	case CHAR:
		status=CA_load_substate_FILE3D(pathToFile,(char*)(substates[substateLabel]),yDim,xDim,zDim);
		break;
	case INT:
		status=CA_load_substate_FILE3D(pathToFile,(int*)(substates[substateLabel]),yDim,xDim,zDim);
		break;
	case BOOL:
		status=CA_load_substate_FILE3D(pathToFile,(bool*)(substates[substateLabel]),yDim,xDim,zDim);
		break;
	}
	return status;
}

int CA3D::saveSubstate(SUBSTATE_LABEL substateLabel, const char* const pathToFile){
	short int status =SUCCESS_OPENING_FILE;
	unsigned int type= substateTypes[substateLabel];
	switch(type){
	case FLOAT:
		status=CA_save_substate_FILE3D(pathToFile,(float*)(substates[substateLabel]),yDim,xDim,zDim);
		break;
	case DOUBLE:
		status=CA_save_substate_FILE3D(pathToFile,(double*)(substates[substateLabel]),yDim,xDim,zDim);
		break;
	case CHAR:
		status=CA_save_substate_FILE3D(pathToFile,(char*)(substates[substateLabel]),yDim,xDim,zDim);
		break;
	case INT:
		status=CA_save_substate_FILE3D(pathToFile,(int*)(substates[substateLabel]),yDim,xDim,zDim);
		break;
	case BOOL:
		status=CA_save_substate_FILE3D(pathToFile,(bool*)(substates[substateLabel]),yDim,xDim,zDim);
		break;

	}
	return status;
}



void CA3D::printSubstate_STDOUT(SUBSTATE_LABEL substateLabel){
	printSubstate_STDOUT(substateLabel,yDim,xDim,zDim);

}

void CA3D::printSubstate_STDOUT(SUBSTATE_LABEL substateLabel, unsigned int YDim, unsigned int XDim, unsigned int ZDim){
	assert(YDim<=yDim && XDim<=xDim );

	unsigned int type= substateTypes[substateLabel];
	switch(type){
	case FLOAT:
		CA_print_STDOUT3D((float*)(substates[substateLabel]),YDim,XDim,ZDim);
		break;
	case DOUBLE:
		CA_print_STDOUT3D((double*)(substates[substateLabel]),YDim,XDim,ZDim);
		break;
	case CHAR:
		CA_print_STDOUT3D((char*)(substates[substateLabel]),YDim,XDim,ZDim);
		break;
	case INT:
		CA_print_STDOUT3D((int*)(substates[substateLabel]),YDim,XDim,ZDim);
		break;
	case BOOL:
		CA_print_STDOUT3D((bool*)(substates[substateLabel]),YDim,XDim,ZDim);
		break;

	}

}


/* ------------------START GET SUBSTATE FAMILY FUNCTION------------------*/
bool CA3D::getSubstateValue_BOOL3D(unsigned int substateLabel,unsigned int i, unsigned int j, unsigned int k) const{
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==BOOL);
	return ((bool*)substates[substateLabel])[getLinearIndex(i,j,k,yDim,xDim,zDim)];
}

double CA3D::getSubstateValue_DOUBLE3D(unsigned int substateLabel,unsigned int i, unsigned int j,unsigned int k)const{
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==DOUBLE);
	return ((double*)substates[substateLabel])[getLinearIndex(i,j,k,yDim,xDim,zDim)];
}

float CA3D::getSubstateValue_FLOAT3D(unsigned int substateLabel,unsigned int i, unsigned int j,unsigned int k)const{
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==FLOAT);
	return ((float*)substates[substateLabel])[getLinearIndex(i,j,k,yDim,xDim,zDim)];
}

int CA3D::getSubstateValue_INT3D(unsigned int substateLabel,unsigned int i, unsigned int j,unsigned int k)const{
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==INT);
	return ((int*)substates[substateLabel])[getLinearIndex(i,j,k,yDim,xDim,zDim)];
}

char CA3D::getSubstateValue_CHAR3D(unsigned int substateLabel,unsigned int i, unsigned int j,unsigned int k)const{
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==CHAR);
	return ((char*)substates[substateLabel])[getLinearIndex(i,j,k,yDim,xDim,zDim)];
}

//mono index cell representation
bool CA3D::getSubstateValue_BOOL3D(unsigned int substateLabel,unsigned int index) const{
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==BOOL);
	return ((bool*)substates[substateLabel])[index];
}

double CA3D::getSubstateValue_DOUBLE3D(unsigned int substateLabel,unsigned int index)const{
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==DOUBLE);
	return ((double*)substates[substateLabel])[index];
}

float CA3D::getSubstateValue_FLOAT3D(unsigned int substateLabel,unsigned int index)const{
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==FLOAT);
	return ((float*)substates[substateLabel])[index];
}

int CA3D::getSubstateValue_INT3D(unsigned int substateLabel,unsigned int index)const{
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==INT);
	return ((int*)substates[substateLabel])[index];
}

char CA3D::getSubstateValue_CHAR3D(unsigned int substateLabel,unsigned int index)const{
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==CHAR);
	return ((char*)substates[substateLabel])[index];
}




/* ------------------END GET SUBSTATE VALUE FAMILY------------------*/


/* ----------------START SET SUBSTATE FAMILY FUNCTION ------------------*/
void CA3D::setSubstateValue_BOOL3D(unsigned int substateLabel,unsigned int i, unsigned int j,unsigned int k,bool const value) {
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==BOOL);
	((bool*)substates[substateLabel])[getLinearIndex(i,j,k,yDim,xDim,zDim)]=value;
}

void CA3D::setSubstateValue_DOUBLE3D(unsigned int substateLabel,unsigned int i, unsigned int j,unsigned int k, double const value){
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==DOUBLE);
	((double*)substates[substateLabel])[getLinearIndex(i,j,k,yDim,xDim,zDim)]=value;
}

void CA3D::setSubstateValue_FLOAT3D(unsigned int substateLabel,unsigned int i, unsigned int j,unsigned int k,float const value){
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==FLOAT);
	((float*)substates[substateLabel])[getLinearIndex(i,j,k,yDim,xDim,zDim)]=value;
}

void CA3D::setSubstateValue_INT3D(unsigned int substateLabel,unsigned int i, unsigned int j,unsigned int k,int const value){
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==INT);
	((int*)substates[substateLabel])[getLinearIndex(i,j,k,yDim,xDim,zDim)]=value;
}

void CA3D::setSubstateValue_CHAR3D(unsigned int substateLabel,unsigned int i, unsigned int j,unsigned int k,char const value){
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==CHAR);
	((char*)substates[substateLabel])[getLinearIndex(i,j,k,yDim,xDim,zDim)]=value;
}


void CA3D::setSubstateValue_BOOL3D(unsigned int substateLabel,unsigned int index,bool const value) {
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==BOOL);
	((bool*)substates[substateLabel])[index]=value;
}

void CA3D::setSubstateValue_DOUBLE3D(unsigned int substateLabel,unsigned int index, double const value){
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==DOUBLE);
	((double*)substates[substateLabel])[index]=value;
}

void CA3D::setSubstateValue_FLOAT3D(unsigned int substateLabel,unsigned int index,float const value){
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==FLOAT);
	((float*)substates[substateLabel])[index]=value;
}

void CA3D::setSubstateValue_INT3D(unsigned int substateLabel,unsigned int index,int const value){
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==INT);
	((int*)substates[substateLabel])[index]=value;
}

void CA3D::setSubstateValue_CHAR3D(unsigned int substateLabel,unsigned int index,char const value){
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==CHAR);
	((char*)substates[substateLabel])[index]=value;
}


/* ------------------END SET SUBSTATE VALUE FAMILY------------------*/



void CA3D::registerStopCondictionCallback(bool(*stopCondition_callback)()){
	assert(stopCondition_callback!=NULL);
	stopCondition=stopCondition_callback;
}



/*It checks whether or not all the callbacks, substates,
 * matrices parameter are in coherent state.
 * If it works correctly computation may take place
 * Return:
 * 		TRUE if everything is OK
 * 		FALSE stop the automata. Finalize memories-> shutdown
 */
//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!IMPLEMENTALA BENEEEEEEEEEEEEEEEEEEEEEEEEEEE
bool CA3D::checkAutomataStatusBeforeComputation(){

	/*at least one substate and one callback have to be provided*/

	/*substate number parameter has to match the substate actually added*/

	/*function callbacks number parameter has to match the callbacks actually registered*/

	/*A stop function has to be provided as callback*/
	return true;
}


void CA3D::globalTransitionFunction_MAINLOOP_callback(){
	clock_t start = clock();


	/*------------------------------------------------------------------------------*/

	unsigned int k=0;
	while(!stop){
		//for each elementary process
		for(k=0;k<elementaryProcesses_size;k++){
			//printf("elementaryProcess -> %i\n",k);
			//loops over all cells of the cellular automata

			(elementaryProcesses[k])<<<dimGrid,blockDim>>>(d_CA);
			hipDeviceSynchronize();


		}
		//printf("DIMGRID(%i,%i,%i), BlockDim(%i,%i,%i)\n",dimGrid.x,dimGrid.y,dimGrid.z,blockDim.x,blockDim.y,blockDim.z);

		steps=steps+1;
		printf("Step = %i\n",steps);
		stop=stopCondition();

		//callback each
		if(steps%stepsBetweenCallback==0){
			//callback occurs
			callback(steps);

		}

	}

	/*-----------------------------------------------------------------------------------*/
	clock_t end = clock();
	elapsedTime = (double)(end - start) / CLOCKS_PER_SEC;
	printf("Step performed = %i\nElapsed Time=%.4f\n",steps,elapsedTime);

}

void CA3D::globalTransitionFunction_MAINLOOP(){
	clock_t start = clock();


	/*------------------------------------------------------------------------------*/

	unsigned int k=0;
	while(!stop){
		//for each elementary process
		for(k=0;k<elementaryProcesses_size;k++){
			//printf("elementaryProcess -> %i\n",k);
			//loops over all cells of the cellular automata

			(elementaryProcesses[k])<<<dimGrid,blockDim>>>(d_CA);
			hipDeviceSynchronize();


		}
		printf("DIMGRID(%i,%i,%i), BlockDim(%i,%i,%i)\n",dimGrid.x,dimGrid.y,dimGrid.z,blockDim.x,blockDim.y,blockDim.z);

		steps=steps+1;
		printf("Step = %i\n",steps);
		stop=stopCondition();


	}

	/*-----------------------------------------------------------------------------------*/
	clock_t end = clock();
	elapsedTime = (double)(end - start) / CLOCKS_PER_SEC;
	printf("Step performed = %i\nElapsed Time=%.4f\n",steps,elapsedTime);

}




void CA3D::globalTransitionFunction(){
	if(!checkAutomataStatusBeforeComputation()){
		//error are printed out by the function checkAutomataStatusBeforeComputation() directly
		cleanup();
		exit(-1);
	}
	globalTransitionFunction_MAINLOOP();
}

void CA3D::registerElementaryProcess( void(*callback)(CA_GPU3D* d_CA ) ){
	assert(callback!=NULL && elementaryProcesses_count < elementaryProcesses_size );
	elementaryProcesses[elementaryProcesses_count]=callback;
	elementaryProcesses_count++;
}


void CA3D::setInitialParameters(unsigned int substates_size,unsigned int transitionFunction_size){
	/**
	 * substates_size = The number of substates of the automaton
	 * transitionFunction_size = The number of transition functions
	 * */
	this->substates_size=substates_size;
	this->elementaryProcesses_size=transitionFunction_size;
}

/**
 * It does preliminary allocation of substates arrays
 * and function callbacks for elementary processes
 * */
void CA3D::initialize(){
	assert(elementaryProcesses==0 && substates==0 && substateTypes==0);
	elementaryProcesses=(void(**)(CA_GPU3D*))malloc(sizeof(void(*)(CA_GPU3D*))*elementaryProcesses_size);
	substates= (void**)malloc(sizeof(void*)*substates_size);
	substateTypes =(TYPE*)malloc(sizeof(TYPE)*substates_size);
}



void CA3D::cleanup(){
	printDebug("CLEANUP - START");
	unsigned int i=0;
	//free all the allocated substates
	for(;i<substate_count;i++){
		if(substates[i]!=0){
			free(substates[i]);
			printDebug("FREED");
		}
	}

	//free(elementaryProcesses);//it is allocated on GPU
	free(substates);
	printDebug("CLEANUP - END");
}

void CA3D::addSubstate(SUBSTATE_LABEL label,TYPE t){

	void * substate=NULL;
	substate=allocateSubstate(t,substate);
	registerSubstate(substate,label,t);

}


void CA3D::registerSubstate(void * buffer,SUBSTATE_LABEL label,TYPE t){
	assert( (substate_count < (substates_size)) && (buffer != NULL) && (label < (substates_size)) );
	substates[label]=buffer;
	substateTypes[label]=t;
	substate_count++;

}

void* CA3D::allocateSubstate(TYPE t,void* buffer){
	switch(t){

	case FLOAT:
		buffer = (float*)malloc(numCells*sizeof(float));
		break;
	case DOUBLE:
		buffer = (double*)malloc(numCells*sizeof(double));
		break;
	case CHAR:
		buffer = (char*)malloc(numCells*sizeof(char));
		break;
	case INT:
		buffer = (int*)malloc(numCells*sizeof(int));
		break;
	case BOOL:
		buffer = (bool*)malloc(numCells*sizeof(bool));
		break;

	}
	//map the correnspondent buffer just created to its type


	return buffer;
}

void CA3D::updateDimGrid(){
	dimGrid.x= (xDim/blockDim.x)+(xDim%blockDim.x == 0 ? 0 : 1);
	dimGrid.y= (yDim/blockDim.y)+(yDim%blockDim.y == 0 ? 0 : 1);
	dimGrid.z= (zDim/blockDim.z)+(zDim%blockDim.z == 0 ? 0 : 1);
}



/*
48  layer -1 layer 0 layer 1
49
50  14 |10 | 17 	 5 | 1 | 8 		23 |19 | 26
51  ---|---|---     ---|---|--- 	---|---|---
52  11 | 9 | 12 	 2 | 0 | 3 		20 |18 | 21
53  ---|---|--- 	---|---|---	 	---|---|---
54  15 |13 | 16 	 6 | 4 | 7 		24 |22 | 25
55  */
unsigned int CA3D::getNeighborIndex3D_MOORE(unsigned int i, unsigned int j,unsigned int k,unsigned int neighbor){
	assert(neighbor<27);
	switch(neighbor){
	//layer 0
	case 0:
		return getLinearIndex(i,j,k,yDim,xDim,zDim);
	case 1:
		return getLinearIndex(i-1,j,k,yDim,xDim,zDim);//one row up
	case 2:
		return getLinearIndex(i,j-1,k,yDim,xDim,zDim);//same row one coloumn left
	case 3:
		return getLinearIndex(i,j+1,k,yDim,xDim,zDim);//same row one coloumn right
	case 4:
		return getLinearIndex(i+1,j,k,yDim,xDim,zDim);//same column one row down
	case 5:
		return getLinearIndex(i-1,j-1,k,yDim,xDim,zDim);//one row up one col left
	case 6:
		return getLinearIndex(i+1,j-1,k,yDim,xDim,zDim);//one row down one col left
	case 7:
		return getLinearIndex(i+1,j+1,k,yDim,xDim,zDim);//row down col right
	case 8:
		return getLinearIndex(i-1,j+1,k,yDim,xDim,zDim);//row up col right

		//layer -1 K=k-1
	case 9:
		return getLinearIndex(i,j,k-1,yDim,xDim,zDim);
	case 10:
		return getLinearIndex(i,j,k-1,yDim,xDim,zDim);//one row up
	case 11:
		return getLinearIndex(i,j-1,k-1,yDim,xDim,zDim);//same row one coloumn left
	case 12:
		return getLinearIndex(i,j+1,k-1,yDim,xDim,zDim);//same row one coloumn right
	case 13:
		return getLinearIndex(i+1,j,k-1,yDim,xDim,zDim);//same column one row down
	case 14:
		return getLinearIndex(i-1,j-1,k-1,yDim,xDim,zDim);//one row up one col left
	case 15:
		return getLinearIndex(i+1,j-1,k-1,yDim,xDim,zDim);//one row down one col left
	case 16:
		return getLinearIndex(i+1,j+1,k-1,yDim,xDim,zDim);//row down col right
	case 17:
		return getLinearIndex(i-1,j+1,k-1,yDim,xDim,zDim);//row up col right
		//layer 1 K=k+1
	case 18:
		return getLinearIndex(i,j,k+1,yDim,xDim,zDim);
	case 19:
		return getLinearIndex(i,j,k+1,yDim,xDim,zDim);//one row up
	case 20:
		return getLinearIndex(i,j-1,k+1,yDim,xDim,zDim);//same row one coloumn left
	case 21:
		return getLinearIndex(i,j+1,k+1,yDim,xDim,zDim);//same row one coloumn right
	case 22:
		return getLinearIndex(i+1,j,k+1,yDim,xDim,zDim);//same column one row down
	case 23:
		return getLinearIndex(i-1,j-1,k+1,yDim,xDim,zDim);//one row up one col left
	case 24:
		return getLinearIndex(i+1,j-1,k+1,yDim,xDim,zDim);//one row down one col left
	case 25:
		return getLinearIndex(i+1,j+1,k+1,yDim,xDim,zDim);//row down col right
	case 26:
		return getLinearIndex(i-1,j+1,k+1,yDim,xDim,zDim);//row up col right

	}

	return NULL;//it should never be executed


}

/*
48  layer -1 layer 0 layer 1
49
50  14 |10 | 17 	 5 | 1 | 8 		23 |19 | 26
51  ---|---|---     ---|---|--- 	---|---|---
52  11 | 9 | 12 	 2 | 0 | 3 		20 |18 | 21
53  ---|---|--- 	---|---|---	 	---|---|---
54  15 |13 | 16 	 6 | 4 | 7 		24 |22 | 25
55  */
unsigned int CA3D::getNeighborIndex3D_MOORE(unsigned int index,unsigned int neighbor){
	assert(neighbor<27);

	switch(neighbor){
	//layer 0
	case 0:
		return index;
	case 1:
		return index-xDim;//one row up
	case 2:
		return index-1;//same row one coloumn left
	case 3:
		return index+1;//same row one coloumn right
	case 4:
		return index+xDim;//same column one row down
	case 5:
		return index-xDim-1;//one row up one col left
	case 6:
		return index+xDim-1;//one row down one col left
	case 7:
		return index+xDim+1;//row down col right
	case 8:
		return index-xDim+1;//row up col right


		//layer -1
	case 9:
		return index + (-(yDim*xDim));
	case 10:
		return index-xDim + (-(yDim*xDim));//one row up
	case 11:
		return index-1 + (-(yDim*xDim));//same row one coloumn left
	case 12:
		return index+1 + (-(yDim*xDim));//same row one coloumn right
	case 13:
		return index+xDim + (-(yDim*xDim));//same column one row down
	case 14:
		return index-xDim-1 + (-(yDim*xDim));//one row up one col left
	case 15:
		return index+xDim-1 + (-(yDim*xDim));//one row down one col left
	case 16:
		return index+xDim+1 + (-(yDim*xDim));//row down col right
	case 17:
		return index-xDim+1 + (-(yDim*xDim));//row up col right



		//layer +1
	case 18:
		return index + ((yDim*xDim));
	case 19:
		return index-xDim + ((yDim*xDim));//one row up
	case 20:
		return index-1 + ((yDim*xDim));//same row one coloumn left
	case 21:
		return index+1 + ((yDim*xDim));//same row one coloumn right
	case 22:
		return index+xDim + ((yDim*xDim));//same column one row down
	case 23:
		return index-xDim-1 + ((yDim*xDim));//one row up one col left
	case 24:
		return index+xDim-1 + ((yDim*xDim));//one row down one col left
	case 25:
		return index+xDim+1 + ((yDim*xDim));//row down col right
	case 26:
		return index-xDim+1 + ((yDim*xDim));//row up col right
	}

	return NULL;//it should never be executed
}





//GETTER AND SETTER
unsigned int CA3D::get_xDim() const {
	return xDim;
}

unsigned int CA3D::getElementaryProcessesSize() const {
	return elementaryProcesses_size;
}

unsigned int CA3D::get_yDim() const {
	return yDim;
}

unsigned int CA3D::getSubstatesSize() const {
	return substates_size;
}



unsigned int CA3D::getBlockdimX() const {
	return blockDim.x;
}

void CA3D::setBlockdimX(unsigned int dimX) {
	if(isPowerOfTwo(dimX)){
		this->blockDim.x = dimX;
	}else{
		printf("WARNING -> setBlockDimX has to be power of 2 -> dimX=%i",DEFAULT_BLOCKDIM_X);
		blockDim.x=DEFAULT_BLOCKDIM_X;
	}
	updateDimGrid();


}

unsigned int CA3D::getBlockDimY() const {
	return this->blockDim.y;
}

void CA3D::setBlockDimY(unsigned int dimY) {
	if(isPowerOfTwo(dimY)){
		this->blockDim.y = dimY;
	}else{
		printf("WARNING -> setBlockDimY has to be power of 2 -> dimY=%i",DEFAULT_BLOCKDIM_Y);
		blockDim.y=DEFAULT_BLOCKDIM_Y;
	}
	updateDimGrid();
}

void CA3D::setBlockdimZ(unsigned int dimZ){
	if(isPowerOfTwo(dimZ)){
		this->blockDim.z = dimZ;
	}else{
		printf("WARNING -> setBlockDimZ has to be power of 2 -> dimZ=%i",DEFAULT_BLOCKDIM_Z);
		blockDim.z=DEFAULT_BLOCKDIM_Z;
	}
	updateDimGrid();
}

unsigned int CA3D::getBlockdimZ() const{
	return this->blockDim.z;
}

unsigned int CA3D::getStepsBetweenCopy() const {
	return stepsBetweenCallback;
}





void CA3D::setStepsBetweenCopy(unsigned int stepsBetweenCopy) {
	this->stepsBetweenCallback = stepsBetweenCopy;
}

unsigned int CA3D::isPowerOfTwo (unsigned int x)
{
	unsigned int powerOfTwo = 1;

	while (powerOfTwo < x && powerOfTwo < 2147483648)
		powerOfTwo *= 2;
	return (x == powerOfTwo);
}


//END OFGETTER AND SETTER

void CA3D::copyBuffersFromGPU(){
	printDebug("START offload copy");
	for(int i=0;i<substates_size;i++){
		copyBufferFromGPU(substates[i],d_subPointer[i],substateTypes[i]);
	}
	printDebug("END offload copy");
}

/**
 * Set default parameters and do normal initialization
 * */
void CA3D::preliminaryCAConstructor() {
	assert(yDim <= 0);
	this->steps=0;
	this->elapsedTime=0.0f;

	substates=NULL;
	substates_size=0;
	substate_count=0;
	substateTypes=0;
	stopCondition=0;
	stop=false;//global transition func main loop ACTIVE

	elementaryProcesses=NULL;
	elementaryProcesses_size=0;
	elementaryProcesses_count=0;

}

//2D constructor
CA3D::CA3D(int YDim,int XDim,int ZDim,bool toroidal){
	this->preliminaryCAConstructor();
	this->yDim=YDim;
	this->xDim=XDim;
	this->zDim=ZDim;
	this->numCells=yDim*xDim*zDim;
	this->isToroidal=toroidal;
	if(isToroidal){
		getLinearIndex=hd_getLinearIndexToroidal3D;
	}else{
		getLinearIndex=hd_getLinearIndexNormal3D;
	}
	blockDim.x=DEFAULT_BLOCKDIM_X;
	blockDim.y=DEFAULT_BLOCKDIM_Y;
	blockDim.z=DEFAULT_BLOCKDIM_Z;
	updateDimGrid();

}




bool CA3D::evolveOneStep() {
	if(!stop){
		for(int k=0;k<elementaryProcesses_size;k++){
			//printf("elementaryProcess -> %i\n",k);
			//loops over all cells of the cellular automata

			(elementaryProcesses[k])<<<dimGrid,blockDim>>>(d_CA);
			hipDeviceSynchronize();


		}
		//printf("DIMGRID(%i,%i,%i), BlockDim(%i,%i,%i)\n",dimGrid.x,dimGrid.y,dimGrid.z,blockDim.x,blockDim.y,blockDim.z);

		steps=steps+1;
		printf("Step = %i\n",steps);
		stop=stopCondition();

		return true;
	}
	return false;
}

bool CA3D::evolveKsteps(unsigned int k) {
	for(int i=0;i<k && !stop ;k++){

		for(k=0;k<elementaryProcesses_size;k++){
			//printf("elementaryProcess -> %i\n",k);
			//loops over all cells of the cellular automata

			(elementaryProcesses[k])<<<dimGrid,blockDim>>>(d_CA);
			hipDeviceSynchronize();
		}
		//printf("DIMGRID(%i,%i,%i), BlockDim(%i,%i,%i)\n",dimGrid.x,dimGrid.y,dimGrid.z,blockDim.x,blockDim.y,blockDim.z);

		steps=steps+1;
		printf("Step = %i\n",steps);
		stop=stopCondition();

		return true;
	}
	return false;
}

void CA3D::setCallback(void(*call)(unsigned int)){
	this->callback=call;
}


