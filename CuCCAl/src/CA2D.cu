#include "hip/hip_runtime.h"
/*
 * CA.cpp
 *
 *  Created on: 21/mar/2014
 *      Author: davide
 */

#include "CA2D.cuh"

#define CUDA_CHECK_RETURN(value) {											\
		hipError_t _m_cudaStat = value;										\
		if (_m_cudaStat != hipSuccess) {										\
			fprintf(stderr, "Error %s at line %d in file %s\n",					\
					hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
					exit(1);															\
		} }

//numcells=yDim*xDim in constructor CA
void* CA2D::allocateGPUBuffer(void * d_buffer,TYPE type){
	switch(type){

	case FLOAT:
		CUDA_CHECK_RETURN(hipMalloc((void**)&d_buffer,sizeof(float)*numCells));
		break;
	case DOUBLE:
		CUDA_CHECK_RETURN(hipMalloc((void**)&d_buffer,sizeof(double)*numCells));
		break;
	case CHAR:
		CUDA_CHECK_RETURN(hipMalloc((void**)&d_buffer,sizeof(char)*numCells));
		break;
	case INT:
		CUDA_CHECK_RETURN(hipMalloc((void**)&d_buffer,sizeof(int)*numCells));
		break;
	case BOOL:
		CUDA_CHECK_RETURN(hipMalloc((void**)&d_buffer,sizeof(bool)*numCells));
		break;

	}

	return d_buffer;
}


void CA2D::copyBufferFromGPU(void* h_to, void* d_from, TYPE type){

	switch(type){

	case FLOAT:
		CUDA_CHECK_RETURN(hipMemcpy(h_to,d_from,sizeof(float)*numCells,hipMemcpyDeviceToHost));
		break;
	case DOUBLE:
		CUDA_CHECK_RETURN(hipMemcpy(h_to,d_from,sizeof(double)*numCells,hipMemcpyDeviceToHost));
		break;
	case CHAR:
		CUDA_CHECK_RETURN(hipMemcpy(h_to,d_from,sizeof(char)*numCells,hipMemcpyDeviceToHost));
		break;
	case INT:
		CUDA_CHECK_RETURN(hipMemcpy(h_to,d_from,sizeof(int)*numCells,hipMemcpyDeviceToHost));
		break;
	case BOOL:
		CUDA_CHECK_RETURN(hipMemcpy(h_to,d_from,sizeof(bool)*numCells,hipMemcpyDeviceToHost));
		break;

	}
}

void CA2D::copyBufferToGPU(void* d_to, void* h_from, TYPE type){

	switch(type){

	case FLOAT:
		CUDA_CHECK_RETURN(hipMemcpy(d_to,h_from,sizeof(float)*numCells,hipMemcpyHostToDevice));
		break;
	case DOUBLE:
		CUDA_CHECK_RETURN(hipMemcpy(d_to,h_from,sizeof(double)*numCells,hipMemcpyHostToDevice));
		break;
	case CHAR:
		CUDA_CHECK_RETURN(hipMemcpy(d_to,h_from,sizeof(char)*numCells,hipMemcpyHostToDevice));
		break;
	case INT:
		CUDA_CHECK_RETURN(hipMemcpy(d_to,h_from,sizeof(int)*numCells,hipMemcpyHostToDevice));
		break;
	case BOOL:
		CUDA_CHECK_RETURN(hipMemcpy(d_to,h_from,sizeof(bool)*numCells,hipMemcpyHostToDevice));
		break;

	}
}

__global__ void initializeDCA(void** d_AllocatedpointerSubstates,TYPE* d_substateTypes,SCALARS_CA_GPU2D* scalarsTOCPY,CA_GPU2D* d_CA){

	d_CA->d_substates=d_AllocatedpointerSubstates;
	d_CA->d_substateTypes=d_substateTypes;
	d_CA->scalars=scalarsTOCPY;


}

__global__ void printValues(CA_GPU2D* d_CA){

	//printf("(%i,%i),(%i,%i)\n",((int*)d_CA->d_substates[0])[threadIdx.x],d_CA->d_substateTypes[0],((int*)d_CA->d_substates[1])[threadIdx.x],d_CA->d_substateTypes[1]);
	//d_CA->d_substates=d_AllocatedpointerSubstates;
	//printf("SCALARS\n yDim=%i, xDim=%i value=%i",d_CA->scalars->yDim,d_CA->scalars->xDim,d_CA->getSubstateValue_INT(Q,threadIdx.x));


}

void CA2D::initializeGPUAutomata(){
	//allocate GPU_CA on GPU
	CUDA_CHECK_RETURN(hipMalloc(&d_CA,sizeof(CA_GPU2D)));
	d_CA_TOCOPY= new CA_GPU2D();

	//cancellaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaa
	//		for(int i=0;i<yDim;i++){
	//			for(int j=0;j<xDim;j++){
	//				if(i%substates[Q]==0){
	//					((bool*)substates[Q])[getLinearIndexNormal(i,j,yDim,xDim)]=true;
	//				((bool*)substates[Q_NEW])[getLinearIndexNormal(i,j,yDim,xDim)]=false;
	//			}
	//		}
	//		}
	//glider
	//	((bool*)substates[Q])[getLinearIndexNormal(5,5,yDim,xDim)]=true;
	//	((bool*)substates[Q])[getLinearIndexNormal(6,5,yDim,xDim)]=true;
	//	((bool*)substates[Q])[getLinearIndexNormal(5,6,yDim,xDim)]=true;
	//	((bool*)substates[Q])[getLinearIndexNormal(6,6,yDim,xDim)]=true;
	//
	//	((bool*)substates[Q])[getLinearIndexNormal(7,7,yDim,xDim)]=true;
	//	((bool*)substates[Q])[getLinearIndexNormal(8,7,yDim,xDim)]=true;
	//	((bool*)substates[Q])[getLinearIndexNormal(7,8,yDim,xDim)]=true;
	//	((bool*)substates[Q])[getLinearIndexNormal(8,8,yDim,xDim)]=true;

	//allocate memory ON GPU

	/*allocate all the substates ON GPU
	substate_size=substate_count=real number of registered buffers -> coherent state of the automata
	checked befor of GPU initialization
	conversion between unsigned int(substateTypes) and TYPE is legal*/
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_CA_TOCOPY->d_substates,sizeof(void*)*substates_size));

	d_subPointer = (void**)malloc(sizeof(void*)*substates_size);
	for(int i=0;i<substates_size;i++){
		d_subPointer[i]=allocateGPUBuffer(d_subPointer[i],(TYPE)substateTypes[i]);
		copyBufferToGPU(d_subPointer[i],substates[i],(TYPE)substateTypes[i]);

	}
	CUDA_CHECK_RETURN(hipMemcpy(d_CA_TOCOPY->d_substates,d_subPointer,sizeof(void*)*substates_size,hipMemcpyHostToDevice));
	//CUDA_CHECK_RETURN(hipFree((void*)(&d_CA_TOCOPY->d_substates[1])));

	//substates type array (allocation and copy, that's a constant array usually)
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_CA_TOCOPY->d_substateTypes,sizeof(TYPE)*substates_size));
	CUDA_CHECK_RETURN(hipMemcpy(d_CA_TOCOPY->d_substateTypes,substateTypes,sizeof(TYPE)*substates_size,hipMemcpyHostToDevice));

	//copyScalars. First create structure to be copied, then allocate memory on GPU->copy structure on GPU->
	//->then link d_CA_TOCOPY->scalars to d_CA->scalars whithin a kernel
	SCALARS_CA_GPU2D* scalars_TOPCOPY = new SCALARS_CA_GPU2D();
	scalars_TOPCOPY->xDim=xDim;
	scalars_TOPCOPY->yDim=yDim;
	scalars_TOPCOPY->stop=stop;
	scalars_TOPCOPY->steps=steps;
	scalars_TOPCOPY->isToroidal=isToroidal;
	scalars_TOPCOPY->substates_size=substates_size;
	scalars_TOPCOPY->numCells=numCells;

	CUDA_CHECK_RETURN(hipMalloc((void**)&d_CA_TOCOPY->scalars,sizeof(SCALARS_CA_GPU2D)));
	CUDA_CHECK_RETURN(hipMemcpy(d_CA_TOCOPY->scalars,scalars_TOPCOPY,sizeof(SCALARS_CA_GPU2D),hipMemcpyHostToDevice));

	free(scalars_TOPCOPY);//not needed anymore

	hipDeviceSynchronize();
	initializeDCA<<<1,1>>>(d_CA_TOCOPY->d_substates,d_CA_TOCOPY->d_substateTypes,d_CA_TOCOPY->scalars,d_CA);


	hipDeviceSynchronize();
	printValues<<<1,10>>>(d_CA);
	hipDeviceSynchronize();



	printDebug("GPU memory allocated");
}

void CA2D::cleanUpGPUAutomata(){
	//FREE MEMORY ON GPU-> remember to free first all the buffers INSIDE the struct
	printDebug("inizio cleanUP GPU");
	for(int i=0;i<substates_size;i++){
		CUDA_CHECK_RETURN(hipFree((void*)((d_subPointer[i]))));

	}
	//free scalars GPU
	CUDA_CHECK_RETURN(hipFree((void*)d_CA_TOCOPY->scalars));
	//CUDA_CHECK_RETURN(hipFree(d_CA));
	free(d_subPointer);
	printDebug("GPU memory freeed");
}

unsigned long long int CA2D::getSteps() const{
	return steps;
}

unsigned int CA2D::getToroidalLinearIndex(unsigned int linearIndex){
	return hd_mod(linearIndex,yDim*xDim);
}

int CA2D::loadSubstate(SUBSTATE_LABEL substateLabel, const char* const pathToFile){
	short int status =SUCCESS_OPENING_FILE;
	unsigned int type= substateTypes[substateLabel];
	switch(type){
	case FLOAT:
		status=CA_load_substate_FILE2D(pathToFile,(float*)(substates[substateLabel]),yDim,xDim);
		break;
	case DOUBLE:
		status=CA_load_substate_FILE2D(pathToFile,(double*)(substates[substateLabel]),yDim,xDim);
		break;
	case CHAR:
		status=CA_load_substate_FILE2D(pathToFile,(char*)(substates[substateLabel]),yDim,xDim);
		break;
	case INT:
		status=CA_load_substate_FILE2D(pathToFile,(int*)(substates[substateLabel]),yDim,xDim);
		break;
	case BOOL:
		status=CA_load_substate_FILE2D(pathToFile,(bool*)(substates[substateLabel]),yDim,xDim);
		break;
	}
	return status;
}

int CA2D::saveSubstate(SUBSTATE_LABEL substateLabel, const char* const pathToFile){
	short int status =SUCCESS_OPENING_FILE;
	unsigned int type= substateTypes[substateLabel];
	switch(type){
	case FLOAT:
		status=CA_save_substate_FILE2D(pathToFile,(float*)(substates[substateLabel]),yDim,xDim);
		break;
	case DOUBLE:
		status=CA_save_substate_FILE2D(pathToFile,(double*)(substates[substateLabel]),yDim,xDim);
		break;
	case CHAR:
		status=CA_save_substate_FILE2D(pathToFile,(char*)(substates[substateLabel]),yDim,xDim);
		break;
	case INT:
		status=CA_save_substate_FILE2D(pathToFile,(int*)(substates[substateLabel]),yDim,xDim);
		break;
	case BOOL:
		status=CA_save_substate_FILE2D(pathToFile,(bool*)(substates[substateLabel]),yDim,xDim);
		break;

	}
	return status;
}



void CA2D::printSubstate_STDOUT(SUBSTATE_LABEL substateLabel){
	printSubstate_STDOUT(substateLabel,yDim,xDim);

}

void CA2D::printSubstate_STDOUT(SUBSTATE_LABEL substateLabel, unsigned int Nrow, unsigned int Ncol){
	assert(Nrow<=yDim && Ncol<=xDim );

	unsigned int type= substateTypes[substateLabel];
	switch(type){
	case FLOAT:
		CA_print_STDOUT2D((float*)(substates[substateLabel]),Nrow,Ncol);
		break;
	case DOUBLE:
		CA_print_STDOUT2D((double*)(substates[substateLabel]),Nrow,Ncol);
		break;
	case CHAR:
		CA_print_STDOUT2D((char*)(substates[substateLabel]),Nrow,Ncol);
		break;
	case INT:
		CA_print_STDOUT2D((int*)(substates[substateLabel]),Nrow,Ncol);
		break;
	case BOOL:
		CA_print_STDOUT2D((bool*)(substates[substateLabel]),Nrow,Ncol);
		break;

	}

}


/* ------------------START GET SUBSTATE FAMILY FUNCTION------------------*/
bool CA2D::getSubstateValue_BOOL(unsigned int substateLabel,unsigned int i, unsigned int j) const{
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==BOOL);
	return ((bool*)substates[substateLabel])[getLinearIndex(i,j,yDim,xDim)];
}

double CA2D::getSubstateValue_DOUBLE(unsigned int substateLabel,unsigned int i, unsigned int j)const{
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==DOUBLE);
	return ((double*)substates[substateLabel])[getLinearIndex(i,j,yDim,xDim)];
}

float CA2D::getSubstateValue_FLOAT(unsigned int substateLabel,unsigned int i, unsigned int j)const{
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==FLOAT);
	return ((float*)substates[substateLabel])[getLinearIndex(i,j,yDim,xDim)];
}

int CA2D::getSubstateValue_INT(unsigned int substateLabel,unsigned int i, unsigned int j)const{
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==INT);
	return ((int*)substates[substateLabel])[getLinearIndex(i,j,yDim,xDim)];
}

char CA2D::getSubstateValue_CHAR(unsigned int substateLabel,unsigned int i, unsigned int j)const{
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==CHAR);
	return ((char*)substates[substateLabel])[getLinearIndex(i,j,yDim,xDim)];
}

//mono index cell representation
bool CA2D::getSubstateValue_BOOL(unsigned int substateLabel,unsigned int index) const{
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==BOOL);
	return ((bool*)substates[substateLabel])[index];
}

double CA2D::getSubstateValue_DOUBLE(unsigned int substateLabel,unsigned int index)const{
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==DOUBLE);
	return ((double*)substates[substateLabel])[index];
}

float CA2D::getSubstateValue_FLOAT(unsigned int substateLabel,unsigned int index)const{
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==FLOAT);
	return ((float*)substates[substateLabel])[index];
}

int CA2D::getSubstateValue_INT(unsigned int substateLabel,unsigned int index)const{
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==INT);
	return ((int*)substates[substateLabel])[index];
}

char CA2D::getSubstateValue_CHAR(unsigned int substateLabel,unsigned int index)const{
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==CHAR);
	return ((char*)substates[substateLabel])[index];
}




/* ------------------END GET SUBSTATE VALUE FAMILY------------------*/


/* ----------------START SET SUBSTATE FAMILY FUNCTION ------------------*/
void CA2D::setSubstateValue2D_BOOL(unsigned int substateLabel,unsigned int i, unsigned int j,bool const value) {
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==BOOL);
	((bool*)substates[substateLabel])[getLinearIndex(i,j,yDim,xDim)]=value;
}

void CA2D::setSubstateValue2D_DOUBLE(unsigned int substateLabel,unsigned int i, unsigned int j, double const value){
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==DOUBLE);
	((double*)substates[substateLabel])[getLinearIndex(i,j,yDim,xDim)]=value;
}

void CA2D::setSubstateValue2D_FLOAT(unsigned int substateLabel,unsigned int i, unsigned int j,float const value){
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==FLOAT);
	((float*)substates[substateLabel])[getLinearIndex(i,j,yDim,xDim)]=value;
}

void CA2D::setSubstateValue2D_INT(unsigned int substateLabel,unsigned int i, unsigned int j,int const value){
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==INT);
	((int*)substates[substateLabel])[getLinearIndex(i,j,yDim,xDim)]=value;
}

void CA2D::setSubstateValue2D_CHAR(unsigned int substateLabel,unsigned int i, unsigned int j,char const value){
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==CHAR);
	((char*)substates[substateLabel])[getLinearIndex(i,j,yDim,xDim)]=value;
}


void CA2D::setSubstateValue_BOOL(unsigned int substateLabel,unsigned int index,bool const value) {
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==BOOL);
	((bool*)substates[substateLabel])[index]=value;
}

void CA2D::setSubstateValue_DOUBLE(unsigned int substateLabel,unsigned int index, double const value){
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==DOUBLE);
	((double*)substates[substateLabel])[index]=value;
}

void CA2D::setSubstateValue_FLOAT(unsigned int substateLabel,unsigned int index,float const value){
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==FLOAT);
	((float*)substates[substateLabel])[index]=value;
}

void CA2D::setSubstateValue_INT(unsigned int substateLabel,unsigned int index,int const value){
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==INT);
	((int*)substates[substateLabel])[index]=value;
}

void CA2D::setSubstateValue_CHAR(unsigned int substateLabel,unsigned int index,char const value){
	assert(substateLabel<=substate_count);
	assert(substateTypes[substateLabel]==CHAR);
	((char*)substates[substateLabel])[index]=value;
}


/* ------------------END SET SUBSTATE VALUE FAMILY------------------*/



void CA2D::registerStopCondictionCallback(bool(*stopCondition_callback)()){
	assert(stopCondition_callback!=NULL);
	stopCondition=stopCondition_callback;
}



/*It checks whether or not all the callbacks, substates,
 * matrices parameter are in coherent state.
 * If it works correctly computation may take place
 * Return:
 * 		TRUE if everything is OK
 * 		FALSE stop the automata. Finalize memories-> shutdown
 */
//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!IMPLEMENTALA BENEEEEEEEEEEEEEEEEEEEEEEEEEEE
bool CA2D::checkAutomataStatusBeforeComputation(){

	/*at least one substate and one callback have to be provided*/

	/*substate number parameter has to match the substate actually added*/

	/*function callbacks number parameter has to match the callbacks actually registered*/

	/*A stop function has to be provided as callback*/
	return true;
}




void CA2D::globalTransitionFunction_MAINLOOP(){
	clock_t start = clock();


	/*------------------------------------------------------------------------------*/

	unsigned int k=0;
	while(!stop){
		//for each elementary process
		for(k=0;k<elementaryProcesses_size;k++){
			//printf("elementaryProcess -> %i\n",k);
			//loops over all cells of the cellular automata

			(elementaryProcesses[k])<<<dimGrid,blockDim>>>(d_CA);
			hipDeviceSynchronize();


		}
		//printf("DIMGRID(%i,%i,%i), BlockDim(%i,%i,%i)\n",dimGrid.x,dimGrid.y,dimGrid.z,blockDim.x,blockDim.y,blockDim.z);

		steps=steps+1;
		printf("Step = %i\n",steps);
		stop=stopCondition();

		//callback each
		if(steps%stepsBetweenCallback==0){
			//callback occurs
			callback(steps);

		}

	}

	/*-----------------------------------------------------------------------------------*/
	clock_t end = clock();
	elapsedTime = (double)(end - start) / CLOCKS_PER_SEC;
	printf("Step performed = %i\nElapsed Time=%.4f\n",steps,elapsedTime);

}

void CA2D::globalTransitionFunction(){
	if(!checkAutomataStatusBeforeComputation()){
		//error are printed out by the function checkAutomataStatusBeforeComputation() directly
		cleanup();
		exit(-1);
	}
	globalTransitionFunction_MAINLOOP();
}

void CA2D::registerElementaryProcess( void(*callback)(CA_GPU2D* d_CA ) ){
	assert(callback!=NULL && elementaryProcesses_count < elementaryProcesses_size );
	elementaryProcesses[elementaryProcesses_count]=callback;
	elementaryProcesses_count++;
}


void CA2D::setInitialParameters(unsigned int substates_size,unsigned int transitionFunction_size){
	/**
	 * substates_size = The number of substates of the automaton
	 * transitionFunction_size = The number of transition functions
	 * */
	this->substates_size=substates_size;
	this->elementaryProcesses_size=transitionFunction_size;
}

/**
 * It does preliminary allocation of substates arrays
 * and function callbacks for elementary processes
 * */
void CA2D::initialize(){
	assert(elementaryProcesses==0 && substates==0 && substateTypes==0);
	elementaryProcesses=(void(**)(CA_GPU2D*))malloc(sizeof(void(*)(CA_GPU2D*))*elementaryProcesses_size);
	substates= (void**)malloc(sizeof(void*)*substates_size);
	substateTypes =(TYPE*)malloc(sizeof(TYPE)*substates_size);
}



void CA2D::cleanup(){
	printDebug("CLEANUP - START");
	unsigned int i=0;
	//free all the allocated substates
	for(;i<substate_count;i++){
		free(substates[i]);
		printDebug("FREED");
	}

	//free(elementaryProcesses);//it is allocated on GPU
	free(substates);
	printDebug("CLEANUP - END");
}

void CA2D::addSubstate(SUBSTATE_LABEL label,TYPE t){

	void * substate=NULL;
	substate=allocateSubstate(t,substate);
	registerSubstate(substate,label,t);

}


void CA2D::registerSubstate(void * buffer,SUBSTATE_LABEL label,TYPE t){
	assert( (substate_count < (substates_size)) && (buffer != NULL) && (label < (substates_size)) );
	substates[label]=buffer;
	substateTypes[label]=t;
	substate_count++;

}

void* CA2D::allocateSubstate(TYPE t,void* buffer){
	switch(t){

	case FLOAT:
		buffer = (float*)malloc(numCells*sizeof(float));
		break;
	case DOUBLE:
		buffer = (double*)malloc(numCells*sizeof(double));
		break;
	case CHAR:
		buffer = (char*)malloc(numCells*sizeof(char));
		break;
	case INT:
		buffer = (int*)malloc(numCells*sizeof(int));
		break;
	case BOOL:
		buffer = (bool*)malloc(numCells*sizeof(bool));
		break;

	}
	//map the correnspondent buffer just created to its type


	return buffer;
}

void CA2D::updateDimGrid(){
	dimGrid.x= (xDim/blockDim.x)+(xDim%blockDim.x == 0 ? 0 : 1);
	dimGrid.y= (yDim/blockDim.y)+(yDim%blockDim.y == 0 ? 0 : 1);
	dimGrid.z=1;//restore default value for 2D automata
}

//2D constructor
CA2D::CA2D(int yDim,int xDim,bool toroidal){
	this->preliminaryCAConstructor();
	this->yDim=yDim;
	this->xDim=xDim;
	this->numCells=yDim*xDim;
	this->isToroidal=toroidal;
	if(isToroidal){
		getLinearIndex=hd_getLinearIndexToroidal2D;
	}else{
		getLinearIndex=hd_getLinearIndexNormal2D;
	}
	blockDim.x=DEFAULT_BLOCKDIM_X;
	blockDim.y=DEFAULT_BLOCKDIM_Y;
	blockDim.z=1;
	updateDimGrid();

}

/*GET i-th NEIGHBOR INDEX functions MOORE NEIGHBORHOOD

	         5 | 1 | 8
	        ---|---|---
	         2 | 0 | 3
	        ---|---|---
	         6 | 4 | 7
 */
unsigned int CA2D::getNeighborIndex2D_MOORE(unsigned int i, unsigned int j,unsigned int neighbor){
	assert(neighbor<9);
	switch(neighbor){
	case 0:
		return getLinearIndex(i,j,yDim,xDim);
	case 1:
		return getLinearIndex(i-1,j,yDim,xDim);//one row up
	case 2:
		return getLinearIndex(i,j-1,yDim,xDim);//same row one coloumn left
	case 3:
		return getLinearIndex(i,j+1,yDim,xDim);//same row one coloumn right
	case 4:
		return getLinearIndex(i+1,j,yDim,xDim);//same column one row down
	case 5:
		return getLinearIndex(i-1,j-1,yDim,xDim);//one row up one col left
	case 6:
		return getLinearIndex(i+1,j-1,yDim,xDim);//one row down one col left
	case 7:
		return getLinearIndex(i+1,j+1,yDim,xDim);//row down col right
	case 8:
		return getLinearIndex(i-1,j+1,yDim,xDim);//row up col right
	}

	return NULL;//it should never be executed


}
unsigned int CA2D::getNeighborIndex2D_MOORE(unsigned int index,unsigned int neighbor){
	assert(neighbor<9);
	switch(neighbor){
	case 0:
		return index;
	case 1:
		return index-xDim;//one row up
	case 2:
		return index-1;//same row one coloumn left
	case 3:
		return index+1;//same row one coloumn right
	case 4:
		return index+xDim;//same column one row down
	case 5:
		return index-xDim-1;//one row up one col left
	case 6:
		return index+xDim-1;//one row down one col left
	case 7:
		return index+xDim+1;//row down col right
	case 8:
		return index-xDim+1;//row up col right
	}

	return NULL;//it should never be executed
}





//GETTER AND SETTER
unsigned int CA2D::get_xDim() const {
	return xDim;
}

unsigned int CA2D::getElementaryProcessesSize() const {
	return elementaryProcesses_size;
}

unsigned int CA2D::get_yDim() const {
	return yDim;
}

unsigned int CA2D::getSubstatesSize() const {
	return substates_size;
}



unsigned int CA2D::getBlockdimX() const {
	return blockDim.x;
}

void CA2D::setBlockdimX(unsigned int dimX) {
	if(isPowerOfTwo(blockDim.x)){
		this->blockDim.x = dimX;
	}else{
		printf("WARNING -> setBlockDimX has to be power of 2 -> dimX=%i",DEFAULT_BLOCKDIM_X);
		blockDim.x=DEFAULT_BLOCKDIM_X;
	}
	updateDimGrid();


}

unsigned int CA2D::getBlockDimY() const {
	return this->blockDim.y;
}

void CA2D::setBlockDimY(unsigned int dimY) {
	if(isPowerOfTwo(blockDim.y)){
		this->blockDim.y = dimY;
	}else{
		printf("WARNING -> setBlockDimY has to be power of 2 -> dimY=%i",DEFAULT_BLOCKDIM_Y);
		blockDim.y=DEFAULT_BLOCKDIM_Y;
	}
	updateDimGrid();
}

unsigned int CA2D::getStepsBetweenCopy() const {
	return stepsBetweenCallback;
}





void CA2D::setStepsBetweenCopy(unsigned int stepsBetweenCopy) {
	this->stepsBetweenCallback = stepsBetweenCopy;
}

unsigned int CA2D::isPowerOfTwo (unsigned int x)
{
	unsigned int powerOfTwo = 1;

	while (powerOfTwo < x && powerOfTwo < 2147483648)
		powerOfTwo *= 2;
	return (x == powerOfTwo);
}


//END OFGETTER AND SETTER

void CA2D::copyBuffersFromGPU(){
	printDebug("START offload copy");
	for(int i=0;i<substates_size;i++){
		copyBufferFromGPU(substates[i],d_subPointer[i],substateTypes[i]);
	}
	printDebug("END offload copy");
}

/**
 * Set default parameters and do normal initialization
 * */
void CA2D::preliminaryCAConstructor() {
	assert(yDim <= 0);
	this->steps=0;
	this->elapsedTime=0.0f;

	substates=NULL;
	substates_size=0;
	substate_count=0;
	substateTypes=0;
	stopCondition=0;
	stop=false;//global transition func main loop ACTIVE

	elementaryProcesses=NULL;
	elementaryProcesses_size=0;
	elementaryProcesses_count=0;

}



void CA2D::setCallback(void(*call)(unsigned int)){
	this->callback=call;
}


