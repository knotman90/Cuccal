#include "hip/hip_runtime.h"
/*
 * config.h
 *
 *  Created on: 20/mar/2014
 *      Author: davide
 */

/*
         5 | 1 | 8
        ---|---|---
         2 | 0 | 3
        ---|---|---
         6 | 4 | 7
 */

#include "config.h"
#include "CA.cuh"
#include <cstring>
#include <iostream>
#include "CAGLVisualizer.h"
//CA dichiarata in CA.h
extern CA CA;


/* Handler for window re-size event. Called back when the window first appears and
   whenever the window is re-sized with its new width and height */
void reshape(GLsizei width, GLsizei height) {  // GLsizei for non-negative integer
	// Compute aspect ratio of the new window
	if (height == 0) height = 1;                // To prevent divide by 0
	GLfloat aspect = (GLfloat)width / (GLfloat)height;

	// Set the viewport to cover the new window
	glViewport(0, 0, width, height);

	// Set the aspect ratio of the clipping area to match the viewport
	glMatrixMode(GL_PROJECTION);  // To operate on the Projection matrix
	glLoadIdentity();             // Reset the projection matrix
	if (width >= height) {
		// aspect >= 1, set the height from -1 to 1, with larger width
		gluOrtho2D(-1.0 * aspect, 1.0 * aspect, -1.0, 1.0);
	} else {
		// aspect < 1, set the width to -1 to 1, with larger height
		gluOrtho2D(-1.0, 1.0, -1.0 / aspect, 1.0 / aspect);
	}
}

//Graphical Callback
void initializeVisualizer(int argc, char** argv){
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB);
	glutInitWindowSize(500,500);
	glutInitWindowPosition(100,100);
	glutCreateWindow("OpenGL - First window demo");
	glutReshapeFunc(reshape);
}



void renderFunction(){
	glClearColor(0.0, 0.0, 0.0, 0.0);
	glClear(GL_COLOR_BUFFER_BIT);
	glColor3f(1.0, 1.0, 1.0);
	glMatrixMode(GL_PROJECTION);      // Select the Projection matrix for operation
	glLoadIdentity();                 // Reset Projection matrix
	gluOrtho2D(0, CA.cols, -300, 0);
	for(float i=0;i<CA.rows;i++){
		for(float j=0;j<CA.cols;j++){
			if(CA.getSubstateValue_BOOL(Q,i,j)){
				glColor3f(1.0, 1.0, 0.0);
			}else{
				glColor3f(1.0, 0.0, 1.0);
			}
			glBegin(GL_POLYGON);
			glVertex2f(j+1, -i+1);
			glVertex2f(j+1, -i);
			glVertex2f(j, -i);
			glVertex2f(j, -i+1);
			glEnd();

		}

	}

	glutSwapBuffers();
	//glutPostRedisplay();
	glFlush();

}






void callback(unsigned int currentsteps){
	char path[20];
	sprintf(path, "Q_%d.sst", currentsteps);
	CA.copyBufferFromGPU(CA.substates[Q],CA.d_subPointer[Q],CA.substateTypes[Q]);
	//glutPostRedisplay();
	//respawn graphicThread and draw substate


	//CA.saveSubstate(Q,path);
}



//mod 2 automaton
//__global__ void gpuEvolve(CA_GPU* d_CA){
//	unsigned int col=(threadIdx.x+blockIdx.x*blockDim.x);
//	unsigned int row=(threadIdx.y+blockIdx.y*blockDim.y);
//	unsigned int totRows=d_CA->scalars->rows;
//	unsigned int totCols=d_CA->scalars->cols;
//	if(row<totRows && col<totCols){
//		short unsigned int count=0;
//		unsigned int linNeighIdx=0;
//		bool alive=d_CA->getSubstateValue_BOOL(Q,row,col);
//		for (int neigh = 1; neigh < 9; neigh++) {
//			linNeighIdx=d_CA->getNeighborIndex_MOORE_Toroidal(row,col,neigh,totRows,totCols);
//			if(d_CA->getSubstateValue_BOOL(Q,linNeighIdx)==true){
//				count++;
//			}
//		}
//		alive=alive%2==0 ? true : false;
//		d_CA->setSubstateValue_BOOL(Q_NEW,row,col,alive);
//	}
//
//}

__global__ void gpuEvolve(CA_GPU* d_CA){
	unsigned int col=(threadIdx.x+blockIdx.x*blockDim.x);
	unsigned int row=(threadIdx.y+blockIdx.y*blockDim.y);
	unsigned int totRows=d_CA->scalars->rows;
	unsigned int totCols=d_CA->scalars->cols;
	if(row<totRows && col<totCols){
		short unsigned int count=0;
		unsigned int linNeighIdx=0;
		bool alive=d_CA->getSubstateValue_BOOL(Q,row,col);
		for (int neigh = 1; neigh < 9; neigh++) {
			linNeighIdx=d_CA->getNeighborIndex_MOORE_Toroidal(row,col,neigh,totRows,totCols);
			if(d_CA->getSubstateValue_BOOL(Q,linNeighIdx)==true){
				count++;
			}
		}
		alive=((!alive && count==3) || (alive && ( count==2 || count==3))) ? true : false;
		d_CA->setSubstateValue_BOOL(Q_NEW,row,col,alive);
	}

}


void __global__ copyBoard(CA_GPU* d_CA){
	int col=(threadIdx.x+blockIdx.x*blockDim.x);
	int row=(threadIdx.y+blockIdx.y*blockDim.y);
	if(row<d_CA->scalars->rows && col<d_CA->scalars->cols){
		d_CA->setSubstateValue_BOOL(Q,row,col,d_CA->getSubstateValue_BOOL(Q_NEW,row,col));
	}

}




//true means --> STOP THE AUTOMATA
bool stopCondition(){

	if(CA.getSteps()>10000){
		return true;
	}
	return false;
}


